
#include "hip/hip_runtime.h"


#include <iostream>
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <algorithm>

/*
TODOs

Wrong result when size != 2^n
Cannot handle (wrong result) when size is huge
*/

using namespace std;

static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		system("pause");
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void GenerateHistogramAndPredicate(int *input, int *currentBit, int *numBits, int *bitHistogram, int *predicate, int *size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= (*size))
	{
		return;
	}

	int bit = (input[id] >> (*currentBit)) & ((1 << *numBits) - 1);

	atomicAdd(&(bitHistogram[bit * gridDim.x + blockIdx.x]), 1);
	predicate[bit * (*size) + id] = 1;
}

__global__ void PrefixSum(int *input, int *output, int *size, int *totalBits)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= (*size))
	{
		return;
	}

	int bit = 0;

	for (bit = 0; bit < (*totalBits); bit++)
	{
		int current_value = input[bit * (*size) + id];
		int current_cdf = input[bit * (*size) + id];

		for (unsigned int interval = 1; interval < blockDim.x; interval <<= 1)
		{
			if (threadIdx.x >= interval)
			{
				current_cdf += input[bit * (*size) + id - interval];
			}
			__syncthreads();

			input[bit * (*size) + id] = current_cdf;
			__syncthreads();
		}

		output[bit * (*size) + id] = input[bit * (*size) + id] - current_value;
		__syncthreads();
	}
}

__global__ void ReOrder(int *input, int *output, int *bitScan, int *relativePos, int *currentBit, int *numBits, int *size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= (*size))
	{
		return;
	}

	int bit = (input[id] >> (*currentBit)) & ((1 << *numBits) - 1);

	output[relativePos[bit * (*size) + id] + bitScan[bit * gridDim.x + blockIdx.x]] = input[id];
}

int pow(int a, int b)
{
	int result = 1;
	for (int i = 0; i < b; i++)
	{
		result *= a;
	}

	return result;
}

const int arraySize = 262144, gridSize = 1024;
const int gridCount = arraySize / gridSize;
int input[arraySize] = { 0 };
int output[arraySize] = { 0 };

int main()
{
	const int totalBits = 22, numBits = 2;
	const int numBitsPow2 = pow(2, numBits);

	int sizeBitScan = numBitsPow2 * gridCount;
	int one = 1;

	int tmp_bitHistogram[32] = { 0 };

	printf("Data generation...\n===============\n| Range: 0 ~ %d\n| Size:  %d\n| GSize: %d\n===============\n\n", pow(2, totalBits), arraySize, gridSize);
	//init data
	srand(time(0));
	for (int i = 0; i < arraySize; i++)
	{
		input[i] = rand() % (pow(2, totalBits) - 1);
	}

	printf("Sending data to GPU...\n");

	//Input:		arraySize				the input array
	//Output:		arraySize				result
	//currentBit:	1						current bit pos
	//bitLenth:		1						current bit lenth (numBits)
	//bitHistogram:	2^numBits				count of items with value i at current bit
	//bitScan:		2^numBits				prefix sum of bitHistogram
	//predicate:	arraySize * 2^numBits	T/F if item value equals to i at current bit
	//relativePos:	arraySize * 2^numBits	prefix sum of predicate
	//size:			1						arraySize
	int *d_Input = 0, *d_Output = 0, *d_bitHistogram = 0, *d_bitScan = 0,
		*d_predicate = 0, *d_relativePos = 0, *d_currentBit = 0, *d_bitLenth = 0, *d_size = 0,
		*d_sizeBitScan = 0, *d_one = 0, *d_bitLenthPow2 = 0;

	// Choose which GPU to run on, change this on a multi-GPU system.
	HANDLE_ERROR(hipSetDevice(0));

	// Allocate GPU buffers
	HANDLE_ERROR(hipMalloc((void**)&d_Output, arraySize * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_Input, arraySize * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_bitHistogram, gridCount * numBitsPow2 * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_bitScan, gridCount * numBitsPow2 * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_predicate, arraySize * numBitsPow2 * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_relativePos, arraySize * numBitsPow2 * sizeof(int)));

	HANDLE_ERROR(hipMalloc((void**)&d_currentBit, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_bitLenth, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_size, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_sizeBitScan, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_one, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_bitLenthPow2, sizeof(int)));

	// Copy input vectors from host memory to GPU buffers.
	HANDLE_ERROR(hipMemcpy(d_Input, input, arraySize * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_bitLenth, &numBits, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_size, &arraySize, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_sizeBitScan, &sizeBitScan, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_bitLenthPow2, &numBitsPow2, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_one, &one, sizeof(int), hipMemcpyHostToDevice));

	printf("GPU Sort Started!\n");

	std::clock_t start;
	start = std::clock();

	//Do the sort
	for (int i = 0; i < totalBits; i += numBits)
	{
		//update current bit
		HANDLE_ERROR(hipMemcpy(d_currentBit, &i, sizeof(int), hipMemcpyHostToDevice));

		//clear buffers
		HANDLE_ERROR(hipMemset(d_bitHistogram, 0, gridCount * numBitsPow2 * sizeof(int)));
		HANDLE_ERROR(hipMemset(d_bitScan, 0, gridCount * numBitsPow2 * sizeof(int)));
		HANDLE_ERROR(hipMemset(d_predicate, 0, numBitsPow2 * arraySize * sizeof(int)));
		HANDLE_ERROR(hipMemset(d_relativePos, 0, numBitsPow2 * arraySize * sizeof(int)));

		////check results
		//HANDLE_ERROR(cudaMemcpy(output, d_Input, arraySize * sizeof(int), cudaMemcpyDeviceToHost));
		//printf("Input:\t");
		//for (int i = 0; i < arraySize; i++)
		//{
		//	printf("%d ", output[i]);
		//}
		//printf("\n");

		/////////////////

		GenerateHistogramAndPredicate <<< gridCount, gridSize >>> (d_Input, d_currentBit, d_bitLenth, d_bitHistogram, d_predicate, d_size);

		////check results
		//HANDLE_ERROR(cudaDeviceSynchronize());
		//HANDLE_ERROR(cudaMemcpy(tmp_bitHistogram, d_bitHistogram, numBitsPow2 * gridCount * sizeof(int), cudaMemcpyDeviceToHost));
		//printf("Bit  %d:\t", i);
		//for (int j = 0; j < gridCount; j++)
		//{
		//	for (int k = 0; k < numBitsPow2; k++)
		//	{
		//		printf("%d ", tmp_bitHistogram[j * numBitsPow2 + k]);
		//	}
		//	printf("| ");
		//}
		//printf("\n");

		/////////////////

		PrefixSum <<< 1, numBitsPow2 * gridCount >>> (d_bitHistogram, d_bitScan, d_sizeBitScan, d_one);

		////check results
		//HANDLE_ERROR(cudaDeviceSynchronize());
		//HANDLE_ERROR(cudaMemcpy(tmp_bitHistogram, d_bitScan, numBitsPow2 * gridCount * sizeof(int), cudaMemcpyDeviceToHost));
		//printf("Scan %d:\t", i);
		//for (int j = 0; j < gridCount; j++)
		//{
		//	for (int k = 0; k < numBitsPow2; k++)
		//	{
		//		printf("%d ", tmp_bitHistogram[j * numBitsPow2 + k]);
		//	}
		//	printf("| ");
		//}
		//printf("\n");

		/////////////////

		////check results
		//HANDLE_ERROR(cudaMemcpy(tmp_bitHistogram, d_predicate, numBitsPow2 * arraySize * sizeof(int), cudaMemcpyDeviceToHost));
		//printf("Pred %d:\t", i);
		//for (int j = 0; j < numBitsPow2; j++)
		//{
		//	for (int k = 0; k < arraySize; k++)
		//	{
		//		printf("%d ", tmp_bitHistogram[j * arraySize + k]);
		//	}
		//	printf("| ");
		//}
		//printf("\n");

		/////////////////

		PrefixSum <<< gridCount, gridSize >>> (d_predicate, d_relativePos, d_size, d_bitLenthPow2);

		////check results
		//HANDLE_ERROR(cudaDeviceSynchronize());
		//HANDLE_ERROR(cudaMemcpy(tmp_bitHistogram, d_relativePos, numBitsPow2 * arraySize * sizeof(int), cudaMemcpyDeviceToHost));
		//printf("RPos %d:\t", i);
		//for (int j = 0; j < numBitsPow2; j++)
		//{
		//	for (int k = 0; k < arraySize; k++)
		//	{
		//		printf("%d ", tmp_bitHistogram[j * arraySize + k]);
		//	}
		//	printf("| ");
		//}
		//printf("\n");

		/////////////////

		ReOrder <<< gridCount, gridSize >>> (d_Input, d_Output, d_bitScan, d_relativePos, d_currentBit, d_bitLenth, d_size);

		////check results
		//HANDLE_ERROR(cudaDeviceSynchronize());
		//HANDLE_ERROR(cudaMemcpy(output, d_Output, arraySize * sizeof(int), cudaMemcpyDeviceToHost));
		//printf("Output:\t");
		//for (int i = 0; i < arraySize; i++)
		//{
		//	printf("%d ", output[i]);
		//}
		//printf("\n");
		HANDLE_ERROR(hipDeviceSynchronize());

		/////////////////
		//Swap input and output for next iter
		int* tmp = d_Input;
		d_Input = d_Output;
		d_Output = tmp;
		
		//printf("\n*-*-*-*-*-*-*\n");
	}
	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	HANDLE_ERROR(hipDeviceSynchronize());

	double duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	printf("\nGPU Sort Finished! time cost (ms): %.3lf\n\n", duration * 1000.0);

	printf("Collecting results...\n");
	HANDLE_ERROR(hipMemcpy(output, d_Input, arraySize * sizeof(int), hipMemcpyDeviceToHost));

	printf("Checking results...\n\n");
	bool validate = true;
	for (int i = 1; i < arraySize; i++)
	{
		if (output[i - 1] > output[i])
		{
			validate = false;
		}
	}

	if (validate)
	{
		printf("Correct!\n");
	}
	else
	{
		printf("Wrong...!\n");
	}

	printf("\n==*==*==*==\nCPU Sort Started!\n");
	start = std::clock();

	std::sort(input, input + arraySize);

	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	printf("\nCPU Sort Finished! time cost (ms): %.3lf\n\n", duration * 1000.0);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
	HANDLE_ERROR(hipDeviceReset());

	hipFree(d_Input);
	hipFree(d_Output);
	hipFree(d_bitHistogram);
	hipFree(d_bitScan);
	hipFree(d_predicate);
	hipFree(d_relativePos);
	hipFree(d_currentBit);
	hipFree(d_bitLenth);
	hipFree(d_size);

	system("pause");

    return 0;
}
